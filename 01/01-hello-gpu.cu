
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

// __global__ keyword indicates that the following function will run on the GPU
__global__ void helloGPU()    
{
  printf("Hello also from the CPU.\n");
}

int main()
{
  helloCPU();

  helloGPU<<<1,1>>>();    // 1 block of 1 thread
  hipDeviceSynchronize();    // will cause the host code (CPU) to wait until the device code (GPU) completes
}
